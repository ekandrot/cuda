#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <time.h>

#define __DRIVER_TYPES_H__
#include "hip/hip_runtime_api.h"

const int LOOP = 1;


const size_t TILE_DIM = 32;
const size_t BLOCK_ROWS = 8;
const size_t totalElements = 256 * 1024 * 1024l;
const size_t totalMemorySize = sizeof(float) * totalElements;

hipEvent_t start, stop;

__global__ void copy1(float *odata, const float *idata) {
    size_t x = blockIdx.x * TILE_DIM + threadIdx.x; // 0..31
    size_t y = blockIdx.y * TILE_DIM + threadIdx.y; // 0..7
    size_t width = gridDim.x * TILE_DIM;

    for (size_t j=0; j<TILE_DIM; j+=BLOCK_ROWS) {
        odata[(y+j)*width + x] = idata[(y+j)*width+x];
    }
}

__global__ void copy2(float *odata, const float *idata) {
    size_t x = blockIdx.x * TILE_DIM + threadIdx.x; // 0..31
    size_t y = blockIdx.y * TILE_DIM + threadIdx.y*4;   // 0,4,8,16,20,24,28
    size_t width = gridDim.x * TILE_DIM;

    for (size_t j=0; j<4; ++j) {
        odata[(y+j)*width + x] = idata[(y+j)*width+x];
    }
}

__global__ void copy3(float *odata, const float *idata) {
    size_t x = blockIdx.x * TILE_DIM + threadIdx.x;
    size_t y = blockIdx.y * TILE_DIM + threadIdx.y;
    size_t width = gridDim.x * TILE_DIM;
    odata[y*width + x] = idata[y*width+x];
}

__global__ void copy3a(float *odata, const float *idata) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // adding this if slows this kernel down to less than copy3,
    // but by a small amount, though makes it work in all memory size cases.
    // it is a trade-off!  If one can guarantee totalElements vs blockDim.x, 
    // then one can get the speed up by removing this check.
    if (tid < totalElements) {
        odata[tid] = idata[tid];
    }
}

__global__ void copy4(float *odata, const float *idata) {
    size_t tid = threadIdx.x + blockDim.x * threadIdx.y;
    tid += blockDim.x * blockDim.y * blockIdx.x;
    size_t width = blockDim.x * blockDim.y * gridDim.x;

    for (size_t j=0; j<totalElements-width; j+=width) {
        odata[tid + j] = idata[tid + j];
    }

    // the final blocks that would overflow
    size_t j=totalElements-width;
    if (tid+j < totalElements) {
        odata[tid + j] = idata[tid + j];
    }
}

float measureCudaMemCpy(float *dev_dst, float *dev_src) {
    float elapsedTimeTotal = 0;
    for (size_t i=0; i<LOOP; ++i) {
        // do a device to device memory copy via kernel, and time it
        checkCudaErrors(hipEventRecord(start, 0));
        checkCudaErrors(hipMemcpy(dev_dst, dev_src, totalMemorySize, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipEventRecord(stop, 0));
        checkCudaErrors(hipEventSynchronize(stop));

        float elapsedTime;
        checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
        elapsedTimeTotal += elapsedTime;
    }

    return elapsedTimeTotal / LOOP;
}

float measureCopy1(float *dev_dst, float *dev_src, dim3 BLOCKS) {
    float elapsedTimeTotal = 0;
    for (size_t i=0; i<LOOP; ++i) {
        // do a device to device memory copy via kernel, and time it
        checkCudaErrors(hipEventRecord(start, 0));
        dim3 THREADS(TILE_DIM, BLOCK_ROWS, 1);
        copy1 <<< BLOCKS, THREADS >>>(dev_dst, dev_src);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipEventRecord(stop, 0));
        checkCudaErrors(hipEventSynchronize(stop));

        float elapsedTime;
        checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
        elapsedTimeTotal += elapsedTime;
    }

    return elapsedTimeTotal / LOOP;
}

float measureCopy2(float *dev_dst, float *dev_src, dim3 BLOCKS) {
    float elapsedTimeTotal = 0;
    for (size_t i=0; i<LOOP; ++i) {
        // do a device to device memory copy via kernel, and time it
        checkCudaErrors(hipEventRecord(start, 0));
        dim3 THREADS(TILE_DIM, BLOCK_ROWS, 1);
        copy2 <<< BLOCKS, THREADS >>>(dev_dst, dev_src);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipEventRecord(stop, 0));
        checkCudaErrors(hipEventSynchronize(stop));

        float elapsedTime;
        checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
        elapsedTimeTotal += elapsedTime;
    }

    return elapsedTimeTotal / LOOP;
}

float measureCopy3(float *dev_dst, float *dev_src, dim3 BLOCKS) {
    float elapsedTimeTotal = 0;
    for (size_t i=0; i<LOOP; ++i) {
        // do a device to device memory copy via kernel, and time it
        checkCudaErrors(hipEventRecord(start, 0));
        dim3 THREADS(TILE_DIM, TILE_DIM, 1);
        copy3 <<< BLOCKS, THREADS >>>(dev_dst, dev_src);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipEventRecord(stop, 0));
        checkCudaErrors(hipEventSynchronize(stop));

        float elapsedTime;
        checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
        elapsedTimeTotal += elapsedTime;
    }

    return elapsedTimeTotal / LOOP;
}

/*
    changed this to use occupancy API.  it calculated the same numbers I had handcoded,
    but this makes it past/future proof.  the copy3a kernel with this code to drive makes
    the least assumptions about the hardware, and is the fastest.

    https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/
    was the article that covered the use of the occupancy api, and the code I copied for debug printing.
*/
float measureCopy3a(float *dev_dst, float *dev_src) {
    int blockSize=0; // to supress warning, assign to zero
    int minGridSize;
    int gridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, copy3a, 0, 0);
    gridSize = (totalElements + blockSize - 1) / blockSize;

    float elapsedTimeTotal = 0;
    for (int i=0; i<LOOP; ++i) {
        // do a device to device memory copy via kernel, and time it
        checkCudaErrors(hipEventRecord(start, 0));
        copy3a <<< gridSize, blockSize >>>(dev_dst, dev_src);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipEventRecord(stop, 0));
        checkCudaErrors(hipEventSynchronize(stop));

        float elapsedTime;
        checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
        elapsedTimeTotal += elapsedTime;
    }

    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, 
                                                 copy3a, blockSize, 
                                                 0);
    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);
    float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / 
                    (float)(props.maxThreadsPerMultiProcessor / 
                            props.warpSize);
    //printf("   [Debugging Info for copy3a] Launched blocks of size %d. Theoretical occupancy: %f\n", blockSize, occupancy);

    return elapsedTimeTotal / LOOP;
}

float measureCopy4(float *dev_dst, float *dev_src, dim3 BLOCKS) {
    float elapsedTimeTotal = 0;
    for (size_t i=0; i<LOOP; ++i) {
        // do a device to device memory copy via kernel, and time it
        checkCudaErrors(hipEventRecord(start, 0));
        dim3 THREADS(TILE_DIM, TILE_DIM, 1);
        copy4 <<< BLOCKS, THREADS >>>(dev_dst, dev_src);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipEventRecord(stop, 0));
        checkCudaErrors(hipEventSynchronize(stop));

        float elapsedTime;
        checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
        elapsedTimeTotal += elapsedTime;
    }

    return elapsedTimeTotal / LOOP;
}

int main(int argc, char **argv) {
    srand(time(NULL));
    checkCudaErrors(hipSetDevice(0));

    float *host_src, *host_dst;
    host_src = (float*)malloc(totalMemorySize);
    host_dst = (float*)malloc(totalMemorySize);

    float *dev_src, *dev_dst;
    checkCudaErrors(hipMalloc((void**)&dev_src, totalMemorySize));
    checkCudaErrors(hipMalloc((void**)&dev_dst, totalMemorySize));

    // fill memory on host side
    float theRand = rand();
    for (size_t i=0; i<totalElements; ++i) {
        host_src[i] = theRand;
    }

    // copy host memory to device
    checkCudaErrors(hipMemcpy(dev_src, host_src, totalMemorySize, hipMemcpyHostToDevice));
    
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    float timePerLoop;


    timePerLoop = measureCudaMemCpy(dev_dst, dev_src);
    printf("mem Elapsed time:  %f s\n", timePerLoop / 1000.0);

    timePerLoop = measureCopy1(dev_dst, dev_src, dim3(totalElements/1024/32/32, 1024, 1));
    printf("1 Elapsed time:  %f s\n", timePerLoop / 1000.0);

    timePerLoop = measureCopy1(dev_dst, dev_src, dim3(totalElements/1024/32/32/32, 1024*32, 1));
    printf("1 Elapsed time:  %f s\n", timePerLoop / 1000.0);

    timePerLoop = measureCopy2(dev_dst, dev_src, dim3(totalElements/1024, 1024/32/32, 1));
    printf("2 Elapsed time:  %f s\n", timePerLoop / 1000.0);

    timePerLoop = measureCopy2(dev_dst, dev_src, dim3(totalElements/1024/32/32, 1024, 1));
    printf("2 Elapsed time:  %f s\n", timePerLoop / 1000.0);

    timePerLoop = measureCopy2(dev_dst, dev_src, dim3(totalElements/1024/32/32/32, 1024*32, 1));
    printf("2 Elapsed time:  %f s\n", timePerLoop / 1000.0);

//    timePerLoop = measureCopy2(dev_dst, dev_src, dim3(totalElements/1024/32/32/64, 1024*64, 1));
//    printf("Elapsed time:  %f s\n", timePerLoop / 1000.0);

    timePerLoop = measureCopy3(dev_dst, dev_src, dim3(totalElements/1024/32/32/32, 1024*32, 1));
    printf("3 Elapsed time:  %f s\n", timePerLoop / 1000.0);

    timePerLoop = measureCopy3a(dev_dst, dev_src);
    printf("3a Elapsed time:  %f s\n", timePerLoop / 1000.0);

    timePerLoop = measureCopy4(dev_dst, dev_src, dim3(26, 1, 1));
    printf("4 Elapsed time:  %f s\n", timePerLoop / 1000.0);

    timePerLoop = measureCopy4(dev_dst, dev_src, dim3(1024, 1, 1));
    printf("4 Elapsed time:  %f s\n", timePerLoop / 1000.0);


    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    checkCudaErrors(hipMemcpy(host_dst, dev_dst, totalMemorySize, hipMemcpyDeviceToHost));
    for (size_t i=0; i<totalElements; ++i) {
        if (host_dst[i] != host_src[i]) {
            printf("*** First mismatch at %ld.  Got %f, was expecting %f  ***\n", i, host_dst[i], host_src[i]);
            break;
        }
    }

    checkCudaErrors(hipFree(dev_src));
    checkCudaErrors(hipFree(dev_dst));
    free(host_src);
    free(host_dst); 

    return 0;
}


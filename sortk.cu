#include "hip/hip_runtime.h"
#include <iostream>

#define __DRIVER_TYPES_H__
#include "hip/hip_runtime_api.h"

const int K = 10;
const int TD = 1;
const int elementsPerRow = 42000;
const int rows = 128;
const int totalElements = elementsPerRow * rows;
const int totalMemorySize = sizeof(float) * totalElements;

/*
*   look for a new home for element i in array a
*   j is the last element of the sorted part of a
*/
__device__ void find_and_insert(float *a, const int i, int j) {
    float t = a[i];
    while (j >= 0 && a[j] > t) {
        a[j+1] = a[j];
        --j;
    }
    a[j+1] = t;
}

__global__ void insertion_sort_k(float *a, const int count, const int k) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    a += tid * count;
    // sort initial k
    for (int i=1; i<k; ++i) {
        if (a[i-1] > a[i]) {
            find_and_insert(a, i, i-1);
        } 
    }

    // find lowest k uing sorted, plus remainder of array
    float maxv = a[k-1];
    int i = k;
    while (i < count) {
#if 0
        float t = a[i];
        int j = k-1;
        while (j >= 0 && a[j] > t) {
            a[j+1] = a[j];
            --j;
        }
        a[j+1] = t;
#else
        if (a[i] < maxv) {
            find_and_insert(a, i, k-2); // -1 for zero-based, -1 because the last will be gone
            maxv = a[k-1]; // actually 5% faster without this line...
        }
#endif
        ++i;
    }
}


int main(int argc, char **argv) {
    checkCudaErrors(hipSetDevice(0));

    float *h_a;
    h_a = (float*)malloc(totalMemorySize);

    for (int i=0; i<totalElements; ++i) {
        h_a[i] = (-i/(float)elementsPerRow)*(i % elementsPerRow);
    }
    //h_a[totalElements-10] = 8.5;

    float *d_a;
    checkCudaErrors(hipMalloc((void**)&d_a, totalMemorySize));
    checkCudaErrors(hipMemcpy(d_a, h_a, totalMemorySize, hipMemcpyHostToDevice));

    dim3 t(TD, 1, 1);
    dim3 b(rows/TD, 1, 1);

    insertion_sort_k<<<b,t>>>(d_a, elementsPerRow, K);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_a, d_a, totalMemorySize, hipMemcpyDeviceToHost));

#if 0
    for (int j=0; j<rows; ++j) {
        for (int i=0; i<K; ++i) {
            std::cout << h_a[i + j*elementsPerRow] << ", ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
#endif

    checkCudaErrors(hipFree(d_a));
    free(h_a);

    return 0;
}

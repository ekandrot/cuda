// Copied from the question at:
// https://stackoverflow.com/questions/5551020/cublas-matrix-multiplication
// then modified so I could compare to hand-coded timings.

#include <stdlib.h>
#include <stdio.h>
#include "hipblas.h"
#define HA 4096
#define WA 4096
#define WB 4096
#define HB WA 
#define WC WB   
#define HC HA  
#define index(i,j,ld) (((j)*(ld))+(i))

#define __DRIVER_TYPES_H__
#include "hip/hip_runtime_api.h"


hipEvent_t start, stop;


void printMat(float*P,int uWP,int uHP){
//printf("\n %f",P[1]);
int i,j;
for(i=0;i<uHP;i++){

    printf("\n");

    for(j=0;j<uWP;j++)
        printf("%f ",P[index(i,j,uHP)]);
        //printf("%f ",P[i*uWP+j]);
}
}




 int  main (int argc, char** argv) {
    hipblasStatus_t status;
        int i,j;
        cublasInit();

        float *A = (float*)malloc(HA*WA*sizeof(float));
        float *B = (float*)malloc(HB*WB*sizeof(float));
        float *C = (float*)malloc(HC*WC*sizeof(float));
    if (A == 0) {
        fprintf (stderr, "!!!! host memory allocation error (A)\n");
        return EXIT_FAILURE;
    }
    if (B == 0) {
        fprintf (stderr, "!!!! host memory allocation error (A)\n");
        return EXIT_FAILURE;
    }
    if (C == 0) {
        fprintf (stderr, "!!!! host memory allocation error (A)\n");
        return EXIT_FAILURE;
        }


    for (i=0;i<HA;i++)
        for (j=0;j<WA;j++)
            A[index(i,j,HA)] = (float) 1;//index(i,j,HA);   
    for (i=0;i<HB;i++)
        for (j=0;j<WB;j++)
            B[index(i,j,HB)] = (float) 1;//index(i,j,HB); 
    /*
    for (i=0;i<HA*WA;i++)
    A[i]=(float) i;
    for (i=0;i<HB*WB;i++)
    B[i]=(float) i;         */  


        float* AA; float* BB; float* CC;

    /*ALLOCATE ON THE DEVICE*/
    status=cublasAlloc(HA*WA,sizeof(float),(void**)&AA);
        if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! device memory allocation error (A)\n");
        return EXIT_FAILURE;
        }

        status=cublasAlloc(HB*WB,sizeof(float),(void**)&BB);
        if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! device memory allocation error (A)\n");
        return EXIT_FAILURE;
        }

        status=cublasAlloc(HC*WC,sizeof(float),(void**)&CC);
        if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! device memory allocation error (A)\n");
        return EXIT_FAILURE;
        }

    /*SET MATRIX*/
        status=hipblasSetMatrix(HA,WA,sizeof(float),A,HA,AA,HA);
        if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! device memory allocation error (A)\n");
        return EXIT_FAILURE;
        }

        status=hipblasSetMatrix(HB,WB,sizeof(float),B,HB,BB,HB);
        if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! device memory allocation error (A)\n");
        return EXIT_FAILURE;
        }

    /*KERNEL*/
    float elapsedTime;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));
        hipblasSgemm('n','n',HA,WB,WA,1,AA,HA,BB,HB,0,CC,HC);
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("cublas Elapsed time:  %f ms\n", elapsedTime);

        status = cublasGetError();
        if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
        }
        hipblasGetMatrix(HC,WC,sizeof(float),CC,HC,C,HC);
        if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! device read error (A)\n");
        return EXIT_FAILURE;
        }


    /* PERFORMANCE OUTPUT*/

/*
    printf("\nMatriz A:\n");
    printMat(A,WA,HA);
    printf("\nMatriz B:\n");
    printMat(B,WB,HB);
    printf("\nMatriz C:\n");
    printMat(C,WC,HC);
*/

    for (int j=0; j<4096; ++j) {
        for (int i=0; i<4096; ++i) {
            if (C[index(i,j,HC)] != 4096.0) {
                printf("Failed at (%d,%d):  %f\n", i, j, C[index(i,j,HC)]);
                goto cleanup;
            }
        }
    }
cleanup:
        free( A );  free( B );  free ( C );
        status = cublasFree(AA);
        if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
        }
        status = cublasFree(BB);
        if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! memory free error (B)\n");
        return EXIT_FAILURE;
        }
        status = cublasFree(CC);
        if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! memory free error (C)\n");
    return EXIT_FAILURE;
    }

        /* Shutdown */
        status = cublasShutdown();
        if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
        }


    return EXIT_SUCCESS; 
}

